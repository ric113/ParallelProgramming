/**********************************************************************
 * DESCRIPTION:
 *   Parallel by CUDA Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

#define MAX_BLOCK_SIZE 1000

void check_param(void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints; 	     		/* total points along string */
float  values[MAXPOINTS+2]; 	/* values at time t */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}
__global__
void kernelWork(float* value_d, int nsteps, int tpoints)
{
        int index = blockIdx.x * 1000 + threadIdx.x;
        float oldvalue, value, newvalue;

        // init .
        float x, fac, k, tmp;

        /* Calculate initial values based on sine curve */
        fac = 2.0 * PI;
        k = index;
        tmp = tpoints - 1;
        x = (float) k / tmp;

        value = sin (fac * x);
        oldvalue = value;

        // update .
        int i;
        float dtime, c, dx, tau, sqtau;

        /* Update values for each time step */
        for (i = 1; i<= nsteps; i++) {
                /* Update points along line for this time step */
                /* global endpoints */
                if ((index  == 0) || (index  == tpoints-1))
                        newvalue = 0.0;
                else {
                        dtime = 0.3;
                        c = 1.0;
                        dx = 1.0;
                        tau = (c * dtime / dx);
                        sqtau = tau * tau;
                        newvalue = (2.0 * value) - oldvalue + (sqtau *  (-2.0)*value);
                }
                /* Update old values with new values */
                oldvalue = value;
                value = newvalue;

        }

        value_d[index+1] = value;

}

void cudaProcess()
{
        float *value_d;

        int float_arr_size = (MAXPOINTS + 2) * sizeof(float);

        hipMalloc( &value_d, float_arr_size );
        hipMemcpy( value_d, values, float_arr_size, hipMemcpyHostToDevice );

        int blockNum = ceil((float)tpoints / MAX_BLOCK_SIZE);

        kernelWork<<<blockNum, MAX_BLOCK_SIZE>>>(value_d, nsteps, tpoints);

        hipMemcpy( values, value_d, float_arr_size, hipMemcpyDeviceToHost );
        hipFree( value_d );
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
	printf("Initializing points on the line...\n");
	printf("Updating all points for all time steps...\n");
  cudaProcess();
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}